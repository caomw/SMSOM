#include "hip/hip_runtime.h"

//cuda5.0
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
//OpenCV
#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
//std
#include <iostream>
#include <vector>
#include <cstdlib>
//
using namespace cv;
using namespace std;

__device__ const float PI = 3.1415926;
__device__ float gaussKernel[3][3] = {1/16.0, 2/16.0, 1/16.0, 2/16.0, 4/16.0, 2/16.0, 1/16.0, 2/16.0, 1/16.0};

__device__ int mi[9] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
__device__ int mj[9] = {0, 0, 0, 1, 1, 1, 2, 2, 2};

__device__ int xlu[9] = {-1, 0, 0, -1, 0, 0, -1, 0, 0};
__device__ int xu[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
__device__ int xru[9] = {0, 0, 1, 0, 0, 1, 0, 0, 1};
__device__ int xr[9] = {0, 0, 1, 0, 0, 1, 0, 0, 1};
__device__ int xrd[9] = {0, 0, 1, 0, 0, 1, 0, 0, 1};
__device__ int xd[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
__device__ int xld[9] = {-1, 0, 0, -1, 0, 0, -1, 0, 0};
__device__ int xl[9] = {-1, 0, 0, -1, 0, 0, -1, 0, 0};

__device__ int ylu[9] = {-1, -1, -1, 0, 0, 0, 0, 0, 0};
__device__ int yu[9] = {-1, -1, -1, 0, 0, 0, 0, 0, 0};
__device__ int yru[9] = {-1, -1, -1, 0, 0, 0, 0, 0, 0};
__device__ int yr[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
__device__ int yrd[9] = {0, 0, 0, 0, 0, 0, 1, 1, 1};
__device__ int yd[9] = {0, 0, 0, 0, 0, 0, 1, 1, 1};
__device__ int yld[9] = {0, 0, 0, 0, 0, 0, 1, 1, 1};
__device__ int yl[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

__device__ float distance(float h1, float s1, float v1, 
	float h2, float s2, float v2){
		return sqrtf(pow(s1*v1*cos(h1*PI/180) - s2*v2*cos(h2*PI/180), 2) +
			pow(s1*v1*sin(h1*PI/180) - s2*v2*sin(h2*PI/180), 2) +
			pow(v1 - v2, 2));
}

__device__ bool shadowRemove(float hi, float si, float vi,
	float hm, float sm, float vm){
		return ( (vi/vm<1) && (vi/vm>0.7) && (si-sm<0.1) && (fabs(hi-hm)<10) );
}

//
__global__ void initLayer(float* input, float* output, int width){
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;//thread index

	for (int j=0; j<3; ++j){
		for (int i=0; i<3; ++i){
			output[(y*3+j)*width*3+(x*3+i)] = input[y*width+x];
		}
	}
}

//foreground detection
__global__ void compete(float* modelH, float* modelS, float* modelV, 
	float* frameH, float* frameS, float* frameV, 
	bool* match, int width){
		int x = blockDim.x*blockIdx.x + threadIdx.x;
		int y = blockDim.y*blockIdx.y + threadIdx.y;

		//used to calculate the distance
		float pointFrame[3];
		float pointModel[9][3];

		pointFrame[0] = frameH[y*width + x];
		pointFrame[1] = frameS[y*width + x];
		pointFrame[2] = frameV[y*width + x];

		for (int j = 0; j < 3; ++j){
			for (int i = 0; i < 3; ++i){
				pointModel[j*3+i][0] = modelH[(y*3+j)*width*3+(x*3+i)];
				pointModel[j*3+i][1] = modelS[(y*3+j)*width*3+(x*3+i)];
				pointModel[j*3+i][2] = modelV[(y*3+j)*width*3+(x*3+i)];
			}
		}

		int index = 0;
		int i2 = 0;
		float min = distance(pointFrame[0], pointFrame[1], pointFrame[2],
			pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

		for (int i2 = 1; i2 < 3*3; ++i2){ 
			float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
				pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
			if (dist <= min){
				min = dist; 
				index = i2;
			}
		}

		for (int j3 = 0; j3 < 3; ++j3){
			for (int i3 = 0; i3 < 3; ++i3){
				match[(y*3+j3)*width*3+(x*3+i3)] = false;
			}
		}
		match[(y*3+mj[index])*width*3+(x*3+mi[index])] = true;
}

__global__ void competeWithFilter(float* model1H, float* model1S, float* model1V,
	float* model2H, float* model2S, float* model2V,
	float* frameH, float* frameS, float* frameV,
	float* maxValue,
	bool* match, int width){
		int x = blockDim.x*blockIdx.x + threadIdx.x;
		int y = blockDim.y*blockIdx.y + threadIdx.y;

		//used to calculate the distance
		float pointFrame[3];
		float pointModel[9][3];

		pointFrame[0] = frameH[y*width + x];
		pointFrame[1] = frameS[y*width + x];
		pointFrame[2] = frameV[y*width + x];

		for (int j = 0; j < 3; ++j){
			for (int i = 0; i < 3; ++i){
				pointModel[j*3+i][0] = model1H[(y*3+j)*width*3+(x*3+i)];
				pointModel[j*3+i][1] = model1S[(y*3+j)*width*3+(x*3+i)];
				pointModel[j*3+i][2] = model1V[(y*3+j)*width*3+(x*3+i)];
			}
		}

		int i2 = 0;
		float max = distance(pointFrame[0], pointFrame[1], pointFrame[2],
			pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

		for (int i2 = 1; i2 < 3*3; ++i2){ 
			float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
				pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
			if (dist >= max)
				max = dist; 
		}

		for (int j3 = 0; j3 < 3; ++j3){
			for (int i3 = 0; i3 < 3; ++i3){
				match[(y*3+j3)*width*3+(x*3+i3)] = false;
			}
		}

		if( max >= maxValue[y*width+x] ){
			for (int j = 0; j < 3; ++j){
				for (int i = 0; i < 3; ++i){
					pointModel[j*3+i][0] = model2H[(y*3+j)*width*3+(x*3+i)];
					pointModel[j*3+i][1] = model2S[(y*3+j)*width*3+(x*3+i)];
					pointModel[j*3+i][2] = model2V[(y*3+j)*width*3+(x*3+i)];
				}
			}

			int index = 0;
			int i2 = 0;
			float min = distance(pointFrame[0], pointFrame[1], pointFrame[2],
				pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

			for (int i2 = 1; i2 < 3*3; ++i2){ 
				float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
					pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
				if (dist <= min){
					min = dist; 
					index = i2;
				}
			}
			match[(y*3+mj[index])*width*3+(x*3+mi[index])] = true;
		}
}

//update the background model
__global__ void cooperate(float* modelH, float* modelS, float* modelV, 
	float* backupH, float* backupS, float* backupV,
	float* frameH, float* frameS, float* frameV,
	bool* match, 
	int width, int height, float alpha){
    int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;

	int m = 0;
	for(int j = 0; j < 3; ++j){
		for(int i = 0; i < 3; ++i){
			m = j*3+i;
			//center
			if(match[(y*3+j)*width*3+(x*3+i)] == true){
				modelH[(y*3+j)*width*3+(x*3+i)] = 
					(1-alpha*gaussKernel[1][1])*backupH[(y*3+j)*width*3+(x*3+i)]
				+ alpha*gaussKernel[1][1]*(frameH[y*width+x]);
				modelS[(y*3+j)*width*3+(x*3+i)] =
					(1-alpha*gaussKernel[1][1])*backupS[(y*3+j)*width*3+(x*3+i)] 
				+ alpha*gaussKernel[1][1]*(frameS[y*width+x]);
				modelV[(y*3+j)*width*3+(x*3+i)] =
					(1-alpha*gaussKernel[1][1])*backupV[(y*3+j)*width*3+(x*3+i)] + 
					alpha*gaussKernel[1][1]*(frameV[y*width+x]);
			}
			//left up
			if (  (x+xlu[m])>=0 && (y+ylu[m])>=0 && 
				match[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i] == true  ){
					modelH[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[2][2])*backupH[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i] 
					+ alpha*gaussKernel[2][2]*(frameH[(y+ylu[m])*width+(x+xlu[m])]);
					modelS[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[2][2])*backupS[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
					+ alpha*gaussKernel[2][2]*(frameS[(y+ylu[m])*width+(x+xlu[m])]);
					modelV[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[2][2])*backupV[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
					+ alpha*gaussKernel[2][2]*(frameV[(y+ylu[m])*width+(x+xlu[m])]);
			}
			//up
			if (  (y+yu[m])>=0 && 
				match[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i] == true  ){
					modelH[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[2][1])*backupH[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
					+ alpha*gaussKernel[2][1]*(frameH[(y+yu[m])*width+(x+xu[m])]);
					modelS[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[2][1])*backupS[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
					+ alpha*gaussKernel[2][1]*(frameS[(y+yu[m])*width+(x+xu[m])]);
					modelV[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[2][1])*backupV[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
					+ alpha*gaussKernel[2][1]*(frameV[(y+yu[m])*width+(x+xu[m])]);
			}
			//right up
			if (  (x+xru[m])<=width && (y+yru[m])>=0 && 
				match[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i] == true  ){
					modelH[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[2][0])*backupH[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
					+ alpha*gaussKernel[2][0]*(frameH[(y+yru[m])*width+(x+xru[m])]);
					modelS[(y*3+j)*width*3+(x*3+i)] =
						(1-alpha*gaussKernel[2][0])*backupS[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i] 
					+ alpha*gaussKernel[2][0]*(frameS[(y+yru[m])*width+(x+xru[m])]);
					modelV[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[2][0])*backupV[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
					+ alpha*gaussKernel[0][2]*(frameV[(y+yru[m])*width+(x+xru[m])]);
			}
			//right
			if (  (x+xr[m])<=width && 
				match[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i] == true  ){
					modelH[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[1][0])*backupH[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
					+ alpha*gaussKernel[1][0]*(frameH[(y+yr[m])*width+(x+xr[m])]);
					modelS[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[1][0])*backupS[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
					+ alpha*gaussKernel[1][0]*(frameS[(y+yr[m])*width+(x+xr[m])]);
					modelV[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[1][0])*backupV[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
					+ alpha*gaussKernel[1][0]*(frameV[(y+yr[m])*width+(x+xr[m])]);
			}
			//right down
			if (  (x+xrd[m])<=width && (y+yrd[m])>=height && 
				match[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i] ==true  ){
					modelH[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[0][0])*backupH[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
					+ alpha*gaussKernel[0][0]*(frameH[(y+yrd[m])*width+(x+xrd[m])]);
					modelS[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[0][0])*backupS[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
					+ alpha*gaussKernel[0][0]*(frameS[(y+yrd[m])*width+(x+xrd[m])]);
					modelV[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[0][0])*backupV[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
					+ alpha*gaussKernel[0][0]*(frameV[(y+yrd[m])*width+(x+xrd[m])]);
			}
			//down
			if (  (y+yd[m])>=height && 
				match[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i] == true  ){
					modelH[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[0][1])*backupH[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
					+ alpha*gaussKernel[0][1]*(frameH[(y+yd[m])*width+(x+xd[m])]);
					modelS[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[0][1])*backupS[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
					+ alpha*gaussKernel[0][1]*(frameS[(y+yd[m])*width+(x+xd[m])]);
					modelV[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[0][1])*backupV[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
					+ alpha*gaussKernel[0][1]*(frameV[(y+yd[m])*width+(x+xd[m])]);
			}
			//left down7
			if (  (y+yld[m])>=height && (x+xld[m])>=0 && 
				match[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i] == true  ){
					modelH[(y*3+j)*width*3+(x*3+i)] =
						(1-alpha*gaussKernel[0][2])*backupH[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
					+ alpha*gaussKernel[0][2]*(frameH[(y+yld[m])*width+(x+xld[m])]);
					modelS[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[0][2])*backupS[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
					+ alpha*gaussKernel[0][2]*(frameS[(y+yld[m])*width+(x+xld[m])]);
					modelV[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[0][2])*backupV[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
					+ alpha*gaussKernel[0][2]*(frameV[(y+yld[m])*width+(x+xld[m])]);
			}
			//left
			if (  (x+xl[m])>=0 && 
				match[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i] == true  ){
					modelH[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[1][2])*backupH[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
					+ alpha*gaussKernel[1][2]*(frameH[(y+yl[m])*width+(x+xl[m])]);
					modelS[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[1][2])*backupS[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
					+ alpha*gaussKernel[1][2]*(frameS[(y+yl[m])*width+(x+xl[m])]);
					modelV[(y*3+j)*width*3+(x*3+i)] = 
						(1-alpha*gaussKernel[1][2])*backupV[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
					+ alpha*gaussKernel[1][2]*(frameV[(y+yl[m])*width+(x+xl[m])]);
			}
		}
	}
}

__global__ void cooperateWithFilter(float* model1H, float* model1S, float* model1V,
	float* model2H, float* model2S, float* model2V, 
	float* backup2H, float* backup2S, float* backup2V,
	float* frameH, float* frameS, float* frameV,
	float* maxValue,
	bool* match, 
	int width, int height, float alpha){
		int x = blockDim.x*blockIdx.x + threadIdx.x;
		int y = blockDim.y*blockIdx.y + threadIdx.y;

		//used to calculate the distance
		float pointFrame[3];
		float pointModel[9][3];

		pointFrame[0] = frameH[y*width + x];
		pointFrame[1] = frameS[y*width + x];
		pointFrame[2] = frameV[y*width + x];

		for (int j = 0; j < 3; ++j){
			for (int i = 0; i < 3; ++i){
				pointModel[j*3+i][0] = model1H[(y*3+j)*width*3+(x*3+i)];
				pointModel[j*3+i][1] = model1S[(y*3+j)*width*3+(x*3+i)];
				pointModel[j*3+i][2] = model1V[(y*3+j)*width*3+(x*3+i)];
			}
		}

		int i2 = 0;
		float max = distance(pointFrame[0], pointFrame[1], pointFrame[2],
			pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

		for (int i2 = 1; i2 < 3*3; ++i2){ 
			float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
				pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
			if (dist >= max)
				max = dist; 
		}

		if( max >= maxValue[y*width+x] ){
			int m = 0;
			for(int j = 0; j < 3; ++j){
				for(int i = 0; i < 3; ++i){
					m = j*3+i;
					//center
					if(match[(y*3+j)*width*3+(x*3+i)] == true){
						model2H[(y*3+j)*width*3+(x*3+i)] = 
							(1-alpha*gaussKernel[1][1])*backup2H[(y*3+j)*width*3+(x*3+i)]
						+ alpha*gaussKernel[1][1]*(frameH[y*width+x]);
						model2S[(y*3+j)*width*3+(x*3+i)] =
							(1-alpha*gaussKernel[1][1])*backup2S[(y*3+j)*width*3+(x*3+i)] 
						+ alpha*gaussKernel[1][1]*(frameS[y*width+x]);
						model2V[(y*3+j)*width*3+(x*3+i)] =
							(1-alpha*gaussKernel[1][1])*backup2V[(y*3+j)*width*3+(x*3+i)] + 
							alpha*gaussKernel[1][1]*(frameV[y*width+x]);
					}
					//left up
					if (  (x+xlu[m])>=0 && (y+ylu[m])>=0 && 
						match[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i] == true  ){
							model2H[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[2][2])*backup2H[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i] 
							+ alpha*gaussKernel[2][2]*(frameH[(y+ylu[m])*width+(x+xlu[m])]);
							model2S[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[2][2])*backup2S[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
							+ alpha*gaussKernel[2][2]*(frameS[(y+ylu[m])*width+(x+xlu[m])]);
							model2V[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[2][2])*backup2V[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
							+ alpha*gaussKernel[2][2]*(frameV[(y+ylu[m])*width+(x+xlu[m])]);
					}
					//up
					if (  (y+yu[m])>=0 && 
						match[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i] == true  ){
							model2H[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[2][1])*backup2H[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
							+ alpha*gaussKernel[2][1]*(frameH[(y+yu[m])*width+(x+xu[m])]);
							model2S[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[2][1])*backup2S[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
							+ alpha*gaussKernel[2][1]*(frameS[(y+yu[m])*width+(x+xu[m])]);
							model2V[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[2][1])*backup2V[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
							+ alpha*gaussKernel[2][1]*(frameV[(y+yu[m])*width+(x+xu[m])]);
					}
					//right up
					if (  (x+xru[m])<=width && (y+yru[m])>=0 && 
						match[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i] == true  ){
							model2H[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[2][0])*backup2H[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
							+ alpha*gaussKernel[2][0]*(frameH[(y+yru[m])*width+(x+xru[m])]);
							model2S[(y*3+j)*width*3+(x*3+i)] =
								(1-alpha*gaussKernel[2][0])*backup2S[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i] 
							+ alpha*gaussKernel[2][0]*(frameS[(y+yru[m])*width+(x+xru[m])]);
							model2V[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[2][0])*backup2V[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
							+ alpha*gaussKernel[0][2]*(frameV[(y+yru[m])*width+(x+xru[m])]);
					}
					//right
					if (  (x+xr[m])<=width && 
						match[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i] == true  ){
							model2H[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[1][0])*backup2H[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
							+ alpha*gaussKernel[1][0]*(frameH[(y+yr[m])*width+(x+xr[m])]);
							model2S[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[1][0])*backup2S[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
							+ alpha*gaussKernel[1][0]*(frameS[(y+yr[m])*width+(x+xr[m])]);
							model2V[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[1][0])*backup2V[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
							+ alpha*gaussKernel[1][0]*(frameV[(y+yr[m])*width+(x+xr[m])]);
					}
					//right down
					if (  (x+xrd[m])<=width && (y+yrd[m])>=height && 
						match[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i] ==true  ){
							model2H[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[0][0])*backup2H[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
							+ alpha*gaussKernel[0][0]*(frameH[(y+yrd[m])*width+(x+xrd[m])]);
							model2S[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[0][0])*backup2S[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
							+ alpha*gaussKernel[0][0]*(frameS[(y+yrd[m])*width+(x+xrd[m])]);
							model2V[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[0][0])*backup2V[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
							+ alpha*gaussKernel[0][0]*(frameV[(y+yrd[m])*width+(x+xrd[m])]);
					}
					//down
					if (  (y+yd[m])>=height && 
						match[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i] == true  ){
							model2H[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[0][1])*backup2H[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
							+ alpha*gaussKernel[0][1]*(frameH[(y+yd[m])*width+(x+xd[m])]);
							model2S[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[0][1])*backup2S[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
							+ alpha*gaussKernel[0][1]*(frameS[(y+yd[m])*width+(x+xd[m])]);
							model2V[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[0][1])*backup2V[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
							+ alpha*gaussKernel[0][1]*(frameV[(y+yd[m])*width+(x+xd[m])]);
					}
					//left down7
					if (  (y+yld[m])>=height && (x+xld[m])>=0 && 
						match[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i] == true  ){
							model2H[(y*3+j)*width*3+(x*3+i)] =
								(1-alpha*gaussKernel[0][2])*backup2H[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
							+ alpha*gaussKernel[0][2]*(frameH[(y+yld[m])*width+(x+xld[m])]);
							model2S[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[0][2])*backup2S[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
							+ alpha*gaussKernel[0][2]*(frameS[(y+yld[m])*width+(x+xld[m])]);
							model2V[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[0][2])*backup2V[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
							+ alpha*gaussKernel[0][2]*(frameV[(y+yld[m])*width+(x+xld[m])]);
					}
					//left
					if (  (x+xl[m])>=0 && 
						match[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i] == true  ){
							model2H[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[1][2])*backup2H[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
							+ alpha*gaussKernel[1][2]*(frameH[(y+yl[m])*width+(x+xl[m])]);
							model2S[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[1][2])*backup2S[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
							+ alpha*gaussKernel[1][2]*(frameS[(y+yl[m])*width+(x+xl[m])]);
							model2V[(y*3+j)*width*3+(x*3+i)] = 
								(1-alpha*gaussKernel[1][2])*backup2V[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
							+ alpha*gaussKernel[1][2]*(frameV[(y+yl[m])*width+(x+xl[m])]);
					}
				}
			}
		}
}

__global__ void initMean(float* modelH, float* modelS, float* modelV, 
	float* frameH, float* frameS, float* frameV,
	float* meanDistance,
	int width){
		int x = blockDim.x*blockIdx.x + threadIdx.x;
		int y = blockDim.y*blockIdx.y + threadIdx.y;

		//used to calculate the distance
		float pointFrame[3];
		float pointModel[9][3];

		pointFrame[0] = frameH[y*width + x];
		pointFrame[1] = frameS[y*width + x];
		pointFrame[2] = frameV[y*width + x];

		for (int j = 0; j < 3; ++j){
			for (int i = 0; i < 3; ++i){
				meanDistance[(y*3+j)*width*3+(x*3+i)] = 
					distance(pointFrame[0], pointFrame[1], pointFrame[2],
					modelH[(y*3+j)*width*3+(x*3+i)], 
					modelS[(y*3+j)*width*3+(x*3+i)],
					modelV[(y*3+j)*width*3+(x*3+i)]);
			}
		}
}

__global__ void meanSum(float* modelH, float* modelS, float* modelV, 
	float* frameH, float* frameS, float* frameV,
	float* meanDistance,
	int width){
		int x = blockDim.x*blockIdx.x + threadIdx.x;
		int y = blockDim.y*blockIdx.y + threadIdx.y;

		//used to calculate the distance
		float pointFrame[3];
		float pointModel[9][3];

		pointFrame[0] = frameH[y*width + x];
		pointFrame[1] = frameS[y*width + x];
		pointFrame[2] = frameV[y*width + x];

		for (int j = 0; j < 3; ++j){
			for (int i = 0; i < 3; ++i){
				meanDistance[(y*3+j)*width*3+(x*3+i)] = (
					meanDistance[(y*3+j)*width*3+(x*3+i)] +
					distance(pointFrame[0], pointFrame[1], pointFrame[2],
					modelH[(y*3+j)*width*3+(x*3+i)], 
					modelS[(y*3+j)*width*3+(x*3+i)],
					modelV[(y*3+j)*width*3+(x*3+i)])
					)/2;
			}
		}
}

__global__ void calculateThreshold(float* meanValue, float* maxValue, float* thresholdValue,
	int width){
		int x = blockDim.x*blockIdx.x + threadIdx.x;
		int y = blockDim.y*blockIdx.y + threadIdx.y;

		float tempMax = meanValue[(y*3)*width*3+(x*3)];
		float tempThreshold = 0;
		for (int j = 0; j < 3; ++j){
			for (int i = 0; i < 3; ++i){
				if( meanValue[(y*3+j)*width*3+(x*3+i)]>=tempMax )
					tempMax = meanValue[(y*3+j)*width*3+(x*3+i)];
				tempThreshold += meanValue[(y*3+j)*width*3+(x*3+i)];
			}
		}

		maxValue[y*width+x] = tempMax;
		thresholdValue[y*width+x] = tempThreshold/9;
}

int thresholdK = 200;
float yipuxilu1 = 0.1;
float yipuxilu2 = 0.03;
float c1 = 1;
float c2 = 0.05;
float alphaLearning = c1*4; // c1/max weight of the Gaussian kernel
float alpha2 = c2*4; // c2/max weight of the Gaussian kernel
int startFrame = 2, endFrame = 799;

int main(){
	char path[200] = "E:\\���ݱ���, ��ǰ���о���\\data�ռ�\\CDnet\\CDnet\\dataset\\dynamicBackground\\overpass\\input\\in%06d.jpg";
	char fileName[200];

	//
	Mat frame;
	sprintf(fileName, path, 1);
	frame = imread(fileName, CV_LOAD_IMAGE_COLOR);
	int width = frame.cols;
	int height = frame.rows;

	Mat frameFloat;
	Mat frameFloat2;
	frameFloat.create(height, width, CV_32FC3);
	frameFloat2.create(height, width, CV_32FC3);
	frame.convertTo(frameFloat, CV_32FC3);
	frameFloat *= 1./255;
	cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);

	vector<Mat> input(3);
	input[0].create(height, width, CV_32FC1);
	input[1].create(height, width, CV_32FC1);
	input[2].create(height, width, CV_32FC1);
	split(frameFloat2, input);

	vector<float*> gpuInput(3);
	vector<float*> gpuLayer1(3);
	vector<float*> gpuLayer1Backup(3);
	bool* gpuMatch1;
	float* gpuOutput;
	float* gpuOutputBackup;

	Mat output;
	output.create(height, width, CV_32FC1);

	for(int i = 0; i < 3; ++i){
		hipMalloc((void**)&gpuInput[i], width*height*sizeof(float));
		hipMalloc((void**)&gpuLayer1[i], width*height*3*3*sizeof(float));
		hipMalloc((void**)&gpuLayer1Backup[i], width*height*3*3*sizeof(float));
		hipMemcpy(gpuInput[i], input[i].data, width*height*sizeof(float), hipMemcpyHostToDevice);
	}
	hipMalloc((void**)&gpuMatch1, width*height*3*3*sizeof(bool));
	hipMalloc((void**)&gpuOutput, width*height*sizeof(float));
	hipMalloc((void**)&gpuOutputBackup, width*height*sizeof(float));

	dim3 grid(width/16, height/16, 1);
	dim3 block(16, 16, 1);

	//Stacked Multi-layer Self Organizing Map (in this code, 2 layers)

	//initialize layer 1
	for(int i = 0; i < 3; ++i){
		initLayer<<<grid, block>>>(gpuInput[i], gpuLayer1[i], width);
	}

	//train layer 1
	cout<<"start training layer 1 ... ..."<<endl;
	for(int i = startFrame; i <= endFrame; ++i){
		if(i%100 == 0)
			cout<<"processing the "<<i<<"th image ... ..."<<endl;
		sprintf(fileName, path, i);
		frame = imread(fileName);
		if(frame.empty()){
			cout<<"There are no images"<<endl;
			return 0;
		}
		frame.convertTo(frameFloat, CV_32FC3);
		frameFloat *= 1./255;
		cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
		split(frameFloat2, input);

		for(int j = 0; j < 3; ++j){
			hipMemcpy(gpuInput[j], input[j].data, width*height*sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(gpuLayer1Backup[j], gpuLayer1[j], width*height*3*3*sizeof(float), hipMemcpyDeviceToDevice);
		}

		compete<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
			gpuInput[0], gpuInput[1], gpuInput[2],
			gpuMatch1, width);
		cooperate<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
			gpuLayer1Backup[0], gpuLayer1Backup[1], gpuLayer1Backup[2],
			gpuInput[0], gpuInput[1], gpuInput[2],
			gpuMatch1,
			width, height, alphaLearning);
	}

	//initialize layer 2
	float* gpuMeanDistance1;
	float* gpuMaxDistance1;
	float* gpuThreshold1;
	hipMalloc((void**)&gpuMeanDistance1, width*height*3*3*sizeof(float));
	hipMalloc((void**)&gpuMaxDistance1, width*height*sizeof(float));
	hipMalloc((void**)&gpuThreshold1, width*height*sizeof(float));
	
	//first frame
	sprintf(fileName, path, 1);
	frame = imread(fileName);
	frame.convertTo(frameFloat, CV_32FC3);
	frameFloat *= 1./255;
	cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
	split(frameFloat2, input);
	for(int i = 0; i < 3; ++i)
		hipMemcpy(gpuInput[i], input[i].data, width*height*sizeof(float), hipMemcpyHostToDevice);
	initMean<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
		gpuInput[0], gpuInput[1], gpuInput[2],
		gpuMeanDistance1, width);

	cout<<"calculate the thresholds for detection and layer 2 input ... ..."<<endl;
	for(int i = startFrame; i <= endFrame; ++i){
		if(i%100 == 0)
			cout<<"processing the "<<i<<"th image ... ..."<<endl;
		sprintf(fileName, path, i);
		frame = imread(fileName);
		if(frame.empty()){
			cout<<"There are no images"<<endl;
			return 0;
		}
		frame.convertTo(frameFloat, CV_32FC3);
		frameFloat *= 1./255;
		cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
		split(frameFloat2, input);

		for(int j = 0; j < 3; ++j){
			hipMemcpy(gpuInput[j], input[j].data, width*height*sizeof(float), hipMemcpyHostToDevice);
		}

		meanSum<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
			gpuInput[0], gpuInput[1], gpuInput[2],
			gpuMeanDistance1, width);
	}

	//
	calculateThreshold<<<grid, block>>>(gpuMeanDistance1, 
		gpuMaxDistance1, gpuThreshold1, width);

	//train layer 2
	vector<float*> gpuLayer2(3);
	vector<float*> gpuLayer2Backup(3);
	bool* gpuMatch2;
	for (int i = 0; i < 3; ++i){
		hipMalloc((void**)&gpuLayer2[i], width*height*3*3*sizeof(float));
		hipMalloc((void**)&gpuLayer2Backup[i], width*height*3*3*sizeof(float));
	}
	hipMalloc((void**)&gpuMatch2, width*height*3*3*sizeof(bool));

	//first frame
	sprintf(fileName, path, 1);
	frame = imread(fileName);
	frame.convertTo(frameFloat, CV_32FC3);
	frameFloat *= 1./255;
	cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
	split(frameFloat2, input);
	for(int i = 0; i < 3; ++i)
		hipMemcpy(gpuInput[i], input[i].data, width*height*sizeof(float), hipMemcpyHostToDevice);
	for(int i = 0; i < 3; ++i){
		initLayer<<<grid, block>>>(gpuInput[i], gpuLayer2[i], width);
	}

	cout<<"start training layer 2 ... ..."<<endl;
	for(int i = startFrame; i <= endFrame; ++i){
		if(i%100 == 0)
			cout<<"processing the "<<i<<"th image ... ..."<<endl;
		sprintf(fileName, path, i);
		frame = imread(fileName);
		if(frame.empty()){
			cout<<"There are no images"<<endl;
			return 0;
		}
		frame.convertTo(frameFloat, CV_32FC3);
		frameFloat *= 1./255;
		cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
		split(frameFloat2, input);

		for(int j = 0; j < 3; ++j){
			hipMemcpy(gpuInput[j], input[j].data, width*height*sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(gpuLayer2Backup[j], gpuLayer2[j], width*height*3*3*sizeof(float), hipMemcpyDeviceToDevice);
		}

		competeWithFilter<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
			gpuLayer2[0], gpuLayer2[1], gpuLayer2[2],
			gpuInput[0], gpuInput[1], gpuInput[2],
			gpuMaxDistance1,
			gpuMatch2, width);
		cooperateWithFilter<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
			gpuLayer2[0], gpuLayer2[1], gpuLayer2[2],
			gpuLayer2Backup[0], gpuLayer2Backup[1], gpuLayer2Backup[2],
			gpuInput[0], gpuInput[1], gpuInput[2],
			gpuMaxDistance1,
			gpuMatch2,
			width, height, alphaLearning);
	}

	for(int i = 0; i < 3; ++i){
		Mat outputTemp;
		outputTemp.create(height*3, width*3, CV_32FC1);
		hipMemcpy(outputTemp.data, gpuLayer2[i], width*height*3*3*sizeof(float), hipMemcpyDeviceToHost);
		namedWindow("layer2", 1);
		imshow("layer2", outputTemp/360);
		waitKey(0);
	}


	//DEBUG
	//hipMemcpy(output.data, gpuMaxDistance1, width*height*sizeof(float), hipMemcpyDeviceToHost);
	//namedWindow("max", 1);
	//imshow("max", output);
	//waitKey(0);

	//hipMemcpy(output.data, gpuThreshold1, width*height*sizeof(float), hipMemcpyDeviceToHost);
	//namedWindow("threshold", 1);
	//imshow("threshold", output);
	//waitKey(0);

	return 0;
}